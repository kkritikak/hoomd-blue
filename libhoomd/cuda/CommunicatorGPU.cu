
#include <hip/hip_runtime.h>
/*
Highly Optimized Object-oriented Many-particle Dynamics -- Blue Edition
(HOOMD-blue) Open Source Software License Copyright 2008-2011 Ames Laboratory
Iowa State University and The Regents of the University of Michigan All rights
reserved.

HOOMD-blue may contain modifications ("Contributions") provided, and to which
copyright is held, by various Contributors who have granted The Regents of the
University of Michigan the right to modify and/or distribute such Contributions.

You may redistribute, use, and create derivate works of HOOMD-blue, in source
and binary forms, provided you abide by the following conditions:

* Redistributions of source code must retain the above copyright notice, this
list of conditions, and the following disclaimer both in the code and
prominently in any materials provided with the distribution.

* Redistributions in binary form must reproduce the above copyright notice, this
list of conditions, and the following disclaimer in the documentation and/or
other materials provided with the distribution.

* All publications and presentations based on HOOMD-blue, including any reports
or published results obtained, in whole or in part, with HOOMD-blue, will
acknowledge its use according to the terms posted at the time of submission on:
http://codeblue.umich.edu/hoomd-blue/citations.html

* Any electronic documents citing HOOMD-Blue will link to the HOOMD-Blue website:
http://codeblue.umich.edu/hoomd-blue/

* Apart from the above required attributions, neither the name of the copyright
holder nor the names of HOOMD-blue's contributors may be used to endorse or
promote products derived from this software without specific prior written
permission.

Disclaimer

THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDER AND CONTRIBUTORS ``AS IS'' AND
ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED
WARRANTIES OF MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE, AND/OR ANY
WARRANTIES THAT THIS SOFTWARE IS FREE OF INFRINGEMENT ARE DISCLAIMED.

IN NO EVENT SHALL THE COPYRIGHT HOLDER OR CONTRIBUTORS BE LIABLE FOR ANY DIRECT,
INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING,
BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE,
DATA, OR PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF
LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN IF
ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
*/

// Maintainer: jglaser

/*! \file CommunicatorGPU.cu
    \brief Implementation of communication algorithms on the GPU
*/

#ifdef ENABLE_MPI
#include "CommunicatorGPU.cuh"
#include "ParticleData.cuh"

#include <thrust/device_vector.h>
#include <thrust/binary_search.h>
#include <thrust/iterator/permutation_iterator.h>
#include <thrust/gather.h>
#include <thrust/scatter.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/copy.h>
#include <thrust/partition.h>
#include <thrust/count.h>

#ifdef WIN32
#include <cassert>
#else
#include <assert.h>
#endif

using namespace thrust;

//! Apply (global) periodic boundary conditions to a ghost particle
struct wrap_ghost_particle
    {
    const gpu_boxsize box;  //!< Dimensions of global simulation box
    const float rghost;     //!< Width of ghost layer
    const unsigned int dir; //!< Direction along which particle was received
    bool is_at_boundary[6]; //!< Flags to indicate whether the local box shares a boundary with the global box

    //! Constructor
    /*! \param _box Dimensions of global simulation box
     * \param _rghost Width of ghost layer
     * \param _dir Direction along which particle was received
     * \param _is_at_boundary Flags to indicate whether the local box shares a boundary with the global box
     */
    wrap_ghost_particle(const gpu_boxsize _box, const float _rghost, const unsigned int _dir, const bool _is_at_boundary[])
        : box(_box), rghost(_rghost), dir(_dir)
        {
        for (unsigned int dir = 0; dir < 6; dir++)
            is_at_boundary[dir] = _is_at_boundary[dir];
        }

    //! Apply peridoic boundary conditions
    /*! \param pos position element to apply boundary conditions to
     * \return the position element with boundary conditions applied
     */
    __host__ __device__ float4 operator()(const float4 &pos)
        {
            // wrap particles received across a global boundary back into global box
            float4 pos2 = pos;
            if (dir==0 && is_at_boundary[1])
                pos2.x -= box.xhi - box.xlo;
            else if (dir==1 && is_at_boundary[0])
                pos2.x += box.xhi - box.xlo;
            else if (dir==2 && is_at_boundary[3])
                pos2.y -= box.yhi - box.ylo;
            else if (dir==3 && is_at_boundary[2])
                pos2.y += box.yhi - box.ylo;
            else if (dir==4 && is_at_boundary[5])
                pos2.z -= box.zhi - box.zlo;
            else if (dir==5 && is_at_boundary[4])
                pos2.z += box.zhi - box.zlo;
            return pos2;
        }
     };

//! Select local particles that within a boundary layer of the neighboring domain in a given direction
struct make_nonbonded_plan : thrust::unary_function<thrust::tuple<float4, unsigned char>, unsigned char>
    {
    const gpu_boxsize box;    //!< Local box dimensions
    const float r_ghost;      //!< Width of boundary layer

    //! Constructor
    /*! \param _box Local box dimensions
     * \param _r_ghost Width of boundary layer
     */
    make_nonbonded_plan(const gpu_boxsize _box, float _r_ghost)
        : box(_box), r_ghost(_r_ghost)
        {
        }

    //! Make exchange plan
    /*! \param t Tuple of Particle position to check and current plan
        \returns The updated plan for this particle
     */
    __host__ __device__ unsigned char operator()(const thrust::tuple<float4, unsigned char>& t)
        {
        float4 pos = thrust::get<0>(t);
        unsigned char plan = thrust::get<1>(t);
        if (pos.x >= box.xhi - r_ghost)
            plan |= send_east;

        if (pos.x < box.xlo + r_ghost)
            plan |= send_west;

        if (pos.y >= box.yhi - r_ghost)
            plan |= send_north;

        if (pos.y < box.ylo + r_ghost)
            plan |= send_south;

        if (pos.z >= box.zhi - r_ghost)
            plan |= send_up;

        if (pos.z < box.zlo + r_ghost)
            plan |= send_down;

        return plan;
        }
     };

//! Select ghost particles for sending in one direction
struct select_particle_ghost
    {
    const unsigned int dir; //!< Current direction

    //! Constructor
    /*! \param _dir Direction of the neighboring domain
     */
    select_particle_ghost(unsigned int _dir)
        : dir(_dir)
        {
        }

    //! Select particles for sending
    /*! \param plan Particle exchange plan
        \returns true if particle is selected for sending
     */
    __host__ __device__ bool operator()(const unsigned char plan)
        {
        return (plan & (1 << dir));
        }
     };


//! Structure to pack a particle data element into
struct __align__(128) pdata_element_gpu
    {
    float4 pos;               //!< Position
    float4 vel;               //!< Velocity
    float3 accel;             //!< Acceleration
    float charge;             //!< Charge
    float diameter;           //!< Diameter
    int3 image;               //!< Image
    unsigned int body;        //!< Body id
    float4 orientation;       //!< Orientation
    unsigned int global_tag;  //!< global tag
    };

//! Get the size of a \c pdata_element_gpu
/*! The CUDA compiler aligns structure members differently than the C++ compiler. This function is used
    to return the actual size as returned by the CUDA compiler.

    \returns the size of a pdata_element_gpu (in bytes)
 */
unsigned int gpu_pdata_element_size()
    {
    return sizeof(pdata_element_gpu);
    }

//! Define a thrust tuple for a particle data element
typedef thrust::tuple<float4,
                      float4,
                      float3,
                      float,
                      float,
                      int3,
                      unsigned int,
                      float4,
                      unsigned int> pdata_tuple_gpu;

//! Select particles to be sent in a specified direction
struct select_particle_migrate_gpu : public thrust::unary_function<const pdata_tuple_gpu&, bool>
    {
    const float xlo;        //!< Lower x boundary
    const float xhi;        //!< Upper x boundary
    const float ylo;        //!< Lower y boundary
    const float yhi;        //!< Upper y boundary
    const float zlo;        //!< Lower z boundary
    const float zhi;        //!< Upper z boundary
    const unsigned int dir; //!< Direction to send particles to
    const float4 *d_pos;    //!< Device array of particle positions


    //! Constructor
    /*!
     */
    select_particle_migrate_gpu(const float _xlo,
                            const float _xhi,
                            const float _ylo,
                            const float _yhi,
                            const float _zlo,
                            const float _zhi,
                            const unsigned int _dir,
                            const float4 *_d_pos)
        : xlo(_xlo), xhi(_xhi), ylo(_ylo), yhi(_yhi), zlo(_zlo), zhi(_zhi), dir(_dir), d_pos(_d_pos)
        {
        }

    //! Select a particle
    /*! t particle data to consider for sending
     * \return true if particle stays in the box
     */
    __host__ __device__ bool operator()(const unsigned int& idx)
        {
        const float4& pos = d_pos[idx];
        // we return true if the particle stays in our box,
        // false otherwise
        return !((dir == 0 && pos.x >= xhi) ||  // send east
                (dir == 1 && pos.x < xlo)  ||  // send west
                (dir == 2 && pos.y >= yhi) ||  // send north
                (dir == 3 && pos.y < ylo)  ||  // send south
                (dir == 4 && pos.z >= zhi) ||  // send up
                (dir == 5 && pos.z < zlo ));   // send down
        }

     };

//! Wrap a received particle across global box boundaries
struct wrap_received_particle
    {
    const gpu_boxsize box;   //!< Dimensions of global simulation box
    const unsigned int dir;  //!< Direction along which the particle was received
    bool is_at_boundary[6]; //!< Flags to indicate whether the local box shares a boundary with the global box

    //! Constructor
    /*! \param _box Dimensions of global simulation box
        \param _dir Direciton along whic the particle was received
        \param _is_at_boundary Flags to indicate whether the local box shares a boundary with the global box
     */
    wrap_received_particle(const gpu_boxsize _box, const unsigned int _dir, const bool _is_at_boundary[])
        : box(_box), dir(_dir)
        {
        for (unsigned int dir = 0; dir < 6; dir++)
            is_at_boundary[dir] = _is_at_boundary[dir];
        }

   //! Wrap particle across boundaries
   /*! \param el particle data element to transform
    * \return transformed particle data element
    */
    __host__ __device__ pdata_element_gpu operator()(const pdata_element_gpu & el)
        {
        pdata_element_gpu el2 = el;
        float4& pos = el2.pos;
        int3& image = el2.image;

        if (dir == 0 && is_at_boundary[1])
            {
            pos.x -= box.xhi - box.xlo;
            image.x++;
            }
        else if (dir == 1 && is_at_boundary[0])
            {
            pos.x += box.xhi - box.xlo;
            image.x--;
            }

        if (dir == 2 && is_at_boundary[3])
            {
            pos.y -= box.yhi - box.ylo;
            image.y++;
            }
        else if (dir == 3 && is_at_boundary[2])
            {
            pos.y += box.yhi - box.ylo;
            image.y--;
            }

        if (dir == 4 && is_at_boundary[5])
            {
            pos.z -= box.zhi - box.zlo;
            image.z++;
            }
        else if (dir == 5 && is_at_boundary[4])
            {
            pos.z += box.zhi - box.zlo;
            image.z--;
            }
        return el2;
        }

     };


//! Determine whether a received particle is to be added to the local box
struct isInBox
    {
    const gpu_boxsize box;  //!< Local box dimensions

    //! Constructor
    /* \param _box Local box dimensions
     */
    isInBox(const gpu_boxsize _box)
        : box(_box)
        {
        }

    //! Determine whether particle is in local box
    /*! \param pos Position of the particle to check
     * \return true if position is in local box
     */
    __host__ __device__ bool check_ptl(const float4& pos)
        {
        return (box.xlo <= pos.x  && pos.x < box.xhi) &&
               (box.ylo <= pos.y  && pos.y < box.yhi) &&
               (box.zlo <= pos.z  && pos.z < box.zhi);
        }

    //! Determine whether particle is in local box
    /*! \param el the particle data element to apply the criterium to
     * \return true if the particle is to be added to the local particle data
     */
    __host__ __device__ bool operator()(const pdata_element_gpu & el)
        {
        return check_ptl(el.pos);
        }

    //! Determine whether particle is in local box
    /*! \param t the particle data tuple to apply the criterium to
     * \return true if the particle is to be added to the local particle data
     */
    __host__ __device__ bool operator()(const pdata_tuple_gpu & t)
        {
        return check_ptl(thrust::get<0>(t));
        }
     };

//! Pack a particle data tuple
struct pack_pdata : public thrust::unary_function<pdata_tuple_gpu, pdata_element_gpu>
    {
    //! Transform operator
    /*! \param t Particle data tuple to pack
     * \return Packed particle data element
     */
    __host__ __device__ pdata_element_gpu operator()(const pdata_tuple_gpu& t)
        {
        pdata_element_gpu el;
        el.pos  = thrust::get<0>(t);
        el.vel  = thrust::get<1>(t);
        el.accel= thrust::get<2>(t);
        el.charge = thrust::get<3>(t);
        el.diameter = thrust::get<4>(t);
        el.image = thrust::get<5>(t);
        el.body = thrust::get<6>(t);
        el.orientation = thrust::get<7>(t);
        el.global_tag = thrust::get<8>(t);
        return el;
        }
    };

//! Unpack a particle data element
struct unpack_pdata : public thrust::unary_function<pdata_element_gpu, pdata_tuple_gpu>
    {
    //! Transform operator
    /*! \param el Particle data element to unpack
     */
    __host__ __device__ pdata_tuple_gpu operator()(const pdata_element_gpu & el)
        {
        return pdata_tuple_gpu(el.pos,
                           el.vel,
                           el.accel,
                           el.charge,
                           el.diameter,
                           el.image,
                           el.body,
                           el.orientation,
                           el.global_tag);
        }
    };

thrust::device_vector<unsigned int> *keys;       //!< Temporary vector of sort keys

void gpu_allocate_tmp_storage()
    {
    keys = new thrust::device_vector<unsigned int>;
    }

void gpu_deallocate_tmp_storage()
    {
    delete keys;
    }

//! GPU Kernel to find incomplete bonds
/*! \param gpu_btable GPU bond table
 * \param pitch Stride of GPU bond table
 * \param n_bonds GPU number of bonds array
 * \param plan Plan array
 * \param d_pos Array of particle positions
 * \param box The local box dimensions
 * \param N number of (local) particles
 */
__global__ void gpu_mark_particles_in_incomplete_bonds_kernel(const uint2 *gpu_btable,
                                                         const unsigned int pitch,
                                                         const unsigned int *n_bonds,
                                                         unsigned char *plan,
                                                         const float4 *d_pos,
                                                         const gpu_boxsize box,
                                                         const unsigned int N)
    {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= N)
        return;

    unsigned int n = n_bonds[idx];
    bool is_complete = true;
    for (unsigned int bond_idx = 0; bond_idx < n; bond_idx++)
        {
        unsigned int idxj = gpu_btable[idx + bond_idx*pitch].x;

        if (! (idxj < N))
            is_complete = false;

        }

    float Lx2 = box.Lx/2.0f;
    float Ly2 = box.Ly/2.0f;
    float Lz2 = box.Lz/2.0f;

    if (! is_complete)
        {
        float4 pos = d_pos[idx];
        unsigned char p = plan[idx];
        p |= (pos.x > box.xlo + Lx2) ? send_east : send_west;
        p |= (pos.y > box.ylo + Ly2) ? send_north : send_south;
        p |= (pos.z > box.zlo + Lz2) ? send_up : send_down;
        plan[idx] = p;
        }
    }

//! Mark particles in incomplete bonds for sending
/* \param d_gpu_btable GPU bond table
 * \param pitch Stride of GPU bond table
 * \param d_n_bonds GPU number of bonds array
 * \param d_plan Plan array
 * \param d_pos Array of particle positions
 * \param box The local box dimensions
 * \param N number of (local) particles
 * \param send_flag Send flag (plan) for ghost particles
 */
void gpu_mark_particles_in_incomplete_bonds(const uint2 *d_gpu_btable,
                                          const unsigned int pitch,
                                          const unsigned int *d_n_bonds,
                                          unsigned char *d_plan,
                                          const float4 *d_pos,
                                          const gpu_boxsize& box,
                                          const unsigned int N)
    {
    assert(d_gpu_btable);
    assert(pitch > 0);
    assert(d_n_bonds);
    assert(d_plan);
    assert(N>0);

    unsigned int block_size = 512;
    gpu_mark_particles_in_incomplete_bonds_kernel<<<N/block_size + 1, block_size>>>(d_gpu_btable,
                                                                                    pitch,
                                                                                    d_n_bonds,
                                                                                    d_plan,
                                                                                    d_pos,
                                                                                    box,
                                                                                    N);
    }

//! Helper kernel to reorder particle data, step one
__global__ void gpu_reorder_pdata_step_one_kernel(const float4 *d_pos,
                                         float4 *d_pos_tmp,
                                         const float4 *d_vel,
                                         float4 *d_vel_tmp,
                                         const float3 *d_accel,
                                         float3 *d_accel_tmp,
                                         const int3 *d_image,
                                         int3 *d_image_tmp,
                                         const float *d_charge,
                                         float *d_charge_tmp,
                                         const float *d_diameter,
                                         float *d_diameter_tmp,
                                         const unsigned int *d_body,
                                         unsigned int *d_body_tmp,
                                         const float4  *d_orientation,
                                         float4 *d_orientation_tmp,
                                         const unsigned int *d_tag,
                                         unsigned int *d_tag_tmp,
                                         unsigned int *keys,
                                         unsigned int N)
    {
    unsigned int idx = blockIdx.x*blockDim.x + threadIdx.x;

    if (idx >= N)
        return;

    unsigned int key = keys[idx];
    d_pos_tmp[idx] = d_pos[key];
    d_vel_tmp[idx] = d_vel[key];
    d_accel_tmp[idx] = d_accel[key];
    d_image_tmp[idx] = d_image[key];
    d_charge_tmp[idx] = d_charge[key];
    d_diameter_tmp[idx] = d_diameter[key];
    d_body_tmp[idx] = d_body[key];
    d_orientation_tmp[idx] = d_orientation[key];
    d_tag_tmp[idx] = d_tag[key];
    }

/*! Reorder the particles according to a migration criterium
 *  Particles that remain in the simulation box come first, followed by the particles that are sent in the
 *  specified direction
 *
 *  \param N Number of particles in local simulation box
 *  \param n_send_ptls Number of particles that are sent (return value)
 *  \param d_pos Array of particle positions
 *  \param d_pos_tmp Array of particle positions to write to
 *  \param d_vel Array of particle velocities
 *  \param d_vel_tmp Array of particle velocities to write to
 *  \param d_accel Array of particle accelerations
 *  \param d_accel_tmp Array of particle accelerations to write to
 *  \param d_image Array of particle images
 *  \param d_image_tmp Array of particle images
 *  \param d_charge Array of particle charges
 *  \param d_charge_tmp Array of particle charges
 *  \param d_diameter Array of particle diameter
 *  \param d_diameter_tmp Array of particle diameter
 *  \param d_body Array of particle body ids
 *  \param d_body_tmp Array of particle body ids
 *  \param d_orientation Array of particle orientations
 *  \param d_orientation_tmp Array of particle orientations
 *  \param d_tag Array of particle global tags
 *  \param d_tag_tmp Array of particle global tags
 *  \param box Dimensions of local simulation box
 *  \param dir Direction to send particles to
 */
void gpu_migrate_select_particles(unsigned int N,
                        unsigned int &n_send_ptls,
                        float4 *d_pos,
                        float4 *d_pos_tmp,
                        float4 *d_vel,
                        float4 *d_vel_tmp,
                        float3 *d_accel,
                        float3 *d_accel_tmp,
                        int3 *d_image,
                        int3 *d_image_tmp,
                        float *d_charge,
                        float *d_charge_tmp,
                        float *d_diameter,
                        float *d_diameter_tmp,
                        unsigned int *d_body,
                        unsigned int *d_body_tmp,
                        float4 *d_orientation,
                        float4 *d_orientation_tmp,
                        unsigned int *d_tag,
                        unsigned int *d_tag_tmp,
                        gpu_boxsize box,
                        unsigned int dir)
    {
    if (keys->size() < N)
        {
        unsigned int cur_size = keys->size() ? keys->size() : N;
        while (cur_size < N) cur_size *= 2;
        keys->resize(cur_size);
        }

    thrust::counting_iterator<unsigned int> count(0);
    thrust::copy(count, count + N, keys->begin());

    thrust::device_vector<unsigned int>::iterator keys_middle;

    keys_middle = thrust::stable_partition(keys->begin(),
                             keys->begin() + N,
                             select_particle_migrate_gpu(box.xlo, box.xhi, box.ylo, box.yhi, box.zlo, box.zhi, dir, d_pos));

    n_send_ptls = (keys->begin() + N) - keys_middle;

    thrust::device_ptr<float4> pos_ptr(d_pos);
    thrust::device_ptr<float4> pos_tmp_ptr(d_pos_tmp);
    thrust::device_ptr<float4> vel_ptr(d_vel);
    thrust::device_ptr<float4> vel_tmp_ptr(d_vel_tmp);
    thrust::device_ptr<float3> accel_ptr(d_accel);
    thrust::device_ptr<float3> accel_tmp_ptr(d_accel_tmp);
    thrust::device_ptr<int3> image_ptr(d_image);
    thrust::device_ptr<int3> image_tmp_ptr(d_image_tmp);
    thrust::device_ptr<float> charge_ptr(d_charge);
    thrust::device_ptr<float> charge_tmp_ptr(d_charge_tmp);
    thrust::device_ptr<float> diameter_ptr(d_diameter);
    thrust::device_ptr<float> diameter_tmp_ptr(d_diameter_tmp);
    thrust::device_ptr<unsigned int> body_ptr(d_body);
    thrust::device_ptr<unsigned int> body_tmp_ptr(d_body_tmp);
    thrust::device_ptr<float4> orientation_ptr(d_orientation);
    thrust::device_ptr<float4> orientation_tmp_ptr(d_orientation_tmp);
    thrust::device_ptr<unsigned int> tag_ptr(d_tag);
    thrust::device_ptr<unsigned int> tag_tmp_ptr(d_tag_tmp);

    // reorder particle data, write into temporary arrays
    thrust::gather(keys->begin(), keys->begin() + N, pos_ptr, pos_tmp_ptr);
    thrust::gather(keys->begin(), keys->begin() + N, vel_ptr, vel_tmp_ptr);
    thrust::gather(keys->begin(), keys->begin() + N, accel_ptr, accel_tmp_ptr);
    thrust::gather(keys->begin(), keys->begin() + N, image_ptr, image_tmp_ptr);
    thrust::gather(keys->begin(), keys->begin() + N, charge_ptr, charge_tmp_ptr);
    thrust::gather(keys->begin(), keys->begin() + N, diameter_ptr, diameter_tmp_ptr);
    thrust::gather(keys->begin(), keys->begin() + N, body_ptr, body_tmp_ptr);
    thrust::gather(keys->begin(), keys->begin() + N, orientation_ptr, orientation_tmp_ptr);
    thrust::gather(keys->begin(), keys->begin() + N, tag_ptr, tag_tmp_ptr);
    }

//! Reset reverse lookup tags of particles we are removing
/* \param n_delete_ptls Number of particles to delete
 * \param d_delete_tags Array of particle tags to delete
 * \param d_rtag Array for tag->idx lookup
 */
void gpu_reset_rtags(unsigned int n_delete_ptls,
                     unsigned int *d_delete_tags,
                     unsigned int *d_rtag)
    {
    thrust::device_ptr<unsigned int> delete_tags_ptr(d_delete_tags);
    thrust::device_ptr<unsigned int> rtag_ptr(d_rtag);

    thrust::constant_iterator<unsigned int> not_local(NOT_LOCAL);
    thrust::scatter(not_local,
                    not_local + n_delete_ptls,
                    delete_tags_ptr,
                    rtag_ptr);
    }

//! Pack particle data into send buffer
/*! \param N number of particles to check for sending
   \param d_pos Array of particle positions
   \param d_vel Array of particle velocities
   \param d_accel Array of particle accelerations
   \param d_image Array of particle images
   \param d_charge Array of particle charges
   \param d_diameter Array of particle diameter
   \param d_body Array of particle body ids
   \param d_orientation Array of particle orientations
   \param d_tag Array of particle global tags
   \param d_send_buf Send buffer (has to be large enough, i.e. maxium size = number of local particles )
   \param d_send_buf_end Pointer to end of send buffer (return value)
*/
void gpu_migrate_pack_send_buffer(unsigned int N,
                           float4 *d_pos,
                           float4 *d_vel,
                           float3 *d_accel,
                           int3 *d_image,
                           float *d_charge,
                           float *d_diameter,
                           unsigned int *d_body,
                           float4  *d_orientation,
                           unsigned int *d_tag,
                           char *d_send_buf,
                           char *&d_send_buf_end)
    {
    thrust::device_ptr<float4> pos_ptr(d_pos);
    thrust::device_ptr<float4> vel_ptr(d_vel);
    thrust::device_ptr<float3> accel_ptr(d_accel);
    thrust::device_ptr<int3> image_ptr(d_image);
    thrust::device_ptr<float> charge_ptr(d_charge);
    thrust::device_ptr<float> diameter_ptr(d_diameter);
    thrust::device_ptr<unsigned int> body_ptr(d_body);
    thrust::device_ptr<float4> orientation_ptr(d_orientation);
    thrust::device_ptr<unsigned int> tag_ptr(d_tag);
    thrust::device_ptr<pdata_element_gpu> send_buf_ptr((pdata_element_gpu *) d_send_buf);

    // we perform operations on the whole particle data
    typedef thrust::tuple<thrust::device_ptr<float4>,
                          thrust::device_ptr<float4>,
                          thrust::device_ptr<float3>,
                          thrust::device_ptr<float>,
                          thrust::device_ptr<float>,
                          thrust::device_ptr<int3>,
                          thrust::device_ptr<unsigned int>,
                          thrust::device_ptr<float4>,
                          thrust::device_ptr<unsigned int> > pdata_iterator_tuple;

    thrust::zip_iterator<pdata_iterator_tuple> pdata_first = thrust::make_tuple( pos_ptr,
                                               vel_ptr,
                                               accel_ptr,
                                               charge_ptr,
                                               diameter_ptr,
                                               image_ptr,
                                               body_ptr,
                                               orientation_ptr,
                                               tag_ptr);
    thrust::zip_iterator<pdata_iterator_tuple> pdata_end = pdata_first + N;


    // pack the particles into the send buffer
    thrust::device_ptr<pdata_element_gpu> send_buf_end_ptr =
        thrust::copy(thrust::make_transform_iterator(pdata_first, pack_pdata()),
                     thrust::make_transform_iterator(pdata_end, pack_pdata()),
                     send_buf_ptr);

    d_send_buf_end = (char *) thrust::raw_pointer_cast(send_buf_end_ptr);
    }

//! Wrap received particles across global box boundaries
/*! \param d_recv_buf Received particle data
 * \param d_recv_buf_end End of received particle data
 * \param n_recv_ptl Number of received particles (return value)
 * \param global_box Dimensions of global box
 * \param dir Direction along which particles where received
 */
void gpu_migrate_wrap_received_particles(char *d_recv_buf,
                                 char *d_recv_buf_end,
                                 unsigned int &n_recv_ptl,
                                 const gpu_boxsize& global_box,
                                 unsigned int dir,
                                 const bool is_at_boundary[])
    {
    thrust::device_ptr<pdata_element_gpu> recv_buf_ptr((pdata_element_gpu *) d_recv_buf);
    thrust::device_ptr<pdata_element_gpu> recv_buf_end_ptr((pdata_element_gpu *) d_recv_buf_end);
    thrust::transform(recv_buf_ptr, recv_buf_end_ptr, recv_buf_ptr, wrap_received_particle(global_box, dir, is_at_boundary));
    n_recv_ptl = recv_buf_end_ptr - recv_buf_ptr;
    }

//! Add received particles to local box if their positions are inside the local boundaries
/*! \param d_recv_buf Buffer of received particle data
 * \param d_recv_buf_end Pointer to end of receive buffer
 * \param d_pos Array to store particle positions
 * \param d_vel Array to store particle velocities
 * \param d_accel Array to store particle accelerations
 * \param d_image Array to store particle images
 * \param d_charge Array to store particle charges
 * \param d_diameter Array to store particle diameters
 * \param d_body Array to store particle body ids
 * \param d_orientation Array to store particle body orientations
 * \param d_tag Array to store particle global tags
 * \param box Local box dimensions
 */
void gpu_migrate_add_particles(  char *d_recv_buf,
                                 char *d_recv_buf_end,
                                 float4 *d_pos,
                                 float4 *d_vel,
                                 float3 *d_accel,
                                 int3 *d_image,
                                 float *d_charge,
                                 float *d_diameter,
                                 unsigned int *d_body,
                                 float4  *d_orientation,
                                 unsigned int *d_tag,
                                 const gpu_boxsize &box)
    {
    thrust::device_ptr<pdata_element_gpu> recv_buf_ptr((pdata_element_gpu *) d_recv_buf);
    thrust::device_ptr<pdata_element_gpu> recv_buf_end_ptr((pdata_element_gpu *) d_recv_buf_end);
    thrust::device_ptr<float4> pos_ptr(d_pos);
    thrust::device_ptr<float4> vel_ptr(d_vel);
    thrust::device_ptr<float3> accel_ptr(d_accel);
    thrust::device_ptr<int3> image_ptr(d_image);
    thrust::device_ptr<float> charge_ptr(d_charge);
    thrust::device_ptr<float> diameter_ptr(d_diameter);
    thrust::device_ptr<unsigned int> body_ptr(d_body);
    thrust::device_ptr<float4> orientation_ptr(d_orientation);
    thrust::device_ptr<unsigned int> tag_ptr(d_tag);

    thrust::copy(thrust::make_transform_iterator(recv_buf_ptr, unpack_pdata()),
                    thrust::make_transform_iterator(recv_buf_end_ptr, unpack_pdata()),
                    make_zip_iterator( thrust::make_tuple( pos_ptr,
                                               vel_ptr,
                                               accel_ptr,
                                               charge_ptr,
                                               diameter_ptr,
                                               image_ptr,
                                               body_ptr,
                                               orientation_ptr,
                                               tag_ptr) )) -
                    make_zip_iterator( thrust::make_tuple( pos_ptr,
                                               vel_ptr,
                                               accel_ptr,
                                               charge_ptr,
                                               diameter_ptr,
                                               image_ptr,
                                               body_ptr,
                                               orientation_ptr,
                                               tag_ptr) );
    }

//! Wrap received ghost particles across global box
/*! \param dir Direction along which particles were received
 * \param n Number of particles to apply periodic boundary conditions to
 * \param d_pos Array of particle positions to apply periodic boundary conditions to
 * \param global_box Dimensions of global simulation box
 * \param rghost Boundary layer width
 */
void gpu_wrap_ghost_particles(unsigned int dir,
                              unsigned int n,
                              float4 *d_pos,
                              gpu_boxsize global_box,
                              float rghost,
                              const bool is_at_boundary[])
    {
    thrust::device_ptr<float4> pos_ptr(d_pos);
    thrust::transform(pos_ptr, pos_ptr +n, pos_ptr, wrap_ghost_particle(global_box, rghost, dir, is_at_boundary));
    }

//! Construct plans for sending non-bonded ghost particles
/*! \param d_plan Array of ghost particle plans
 * \param N number of particles to check
 * \param d_pos Array of particle positions
 * \param box Dimensions of local simulation box
 * \param r_ghost Width of boundary layer
 */
void gpu_make_nonbonded_exchange_plan(unsigned char *d_plan,
                                      unsigned int N,
                                      float4 *d_pos,
                                      gpu_boxsize box,
                                      float r_ghost)
    {
    thrust::device_ptr<float4> pos_ptr(d_pos);
    thrust::device_ptr<unsigned char> plan_ptr(d_plan);

    thrust::transform(
        thrust::make_zip_iterator(thrust::make_tuple(
            pos_ptr,
            plan_ptr)),
        thrust::make_zip_iterator(thrust::make_tuple(
            pos_ptr,
            plan_ptr)) + N,
        plan_ptr,
        make_nonbonded_plan(box, r_ghost));
    }

//! Construct a list of particle tags to send as ghost particles
/*! \param N number of particles to check
 * \param dir Direction in which ghost particles are sent
 * \param d_plan Array of particle exchange plans
 * \param d_global_tag Array of particle global tags
 * \param d_copy_ghosts Array to be fillled x with global tags of particles that are to be send as ghosts
 * \param n_copy_ghosts Number of local particles that are sent in the given direction as ghosts (return value)
 */
void gpu_make_exchange_ghost_list(unsigned int N,
                                  unsigned int dir,
                                  unsigned char *d_plan,
                                  unsigned int *d_global_tag,
                                  unsigned int* d_copy_ghosts,
                                  unsigned int &n_copy_ghosts)
    {
    thrust::device_ptr<unsigned char> plan_ptr(d_plan);
    thrust::device_ptr<unsigned int> global_tag_ptr(d_global_tag);
    thrust::device_ptr<unsigned int> copy_ghosts_ptr(d_copy_ghosts);

    thrust::device_ptr<unsigned int> copy_ghosts_end_ptr;

    copy_ghosts_end_ptr = thrust::copy_if(global_tag_ptr,
                                          global_tag_ptr+N,
                                          plan_ptr,
                                          copy_ghosts_ptr,
                                          select_particle_ghost(dir));

    n_copy_ghosts =  copy_ghosts_end_ptr - copy_ghosts_ptr;
    }

//! Fill send buffers of particles we are sending as ghost particles with partial particle data
/*! \param nghost Number of ghost particles to copy into send buffers
 * \param d_copy_ghosts Array of particle tags to copy as ghost particles
 * \param d_rtag Inverse look-up array for global tags <-> local indices
 * \param d_pos Array of particle positions
 * \param d_pos_copybuf Send buffer for particle positions
 * \param d_charge Array of particle charges
 * \param d_charge_copybuf Send buffer for particle charges
 * \param d_diameter Array of particle diameters
 * \param d_diameter_copybuf Send buffer for particle diameters
 * \param d_plan Array of particle plans
 * \param d_plan_copybuf Send buffer for particle plans
 */
void gpu_exchange_ghosts(unsigned int nghost,
                         unsigned int *d_copy_ghosts,
                         unsigned int *d_rtag,
                         float4 *d_pos,
                         float4 *d_pos_copybuf,
                         float *d_charge,
                         float *d_charge_copybuf,
                         float *d_diameter,
                         float *d_diameter_copybuf,
                         unsigned char *d_plan,
                         unsigned char *d_plan_copybuf)
    {
    thrust::device_ptr<unsigned int> copy_ghosts_ptr(d_copy_ghosts);
    thrust::device_ptr<unsigned int> rtag_ptr(d_rtag);
    thrust::device_ptr<float4> pos_ptr(d_pos);
    thrust::device_ptr<float4> pos_copybuf_ptr(d_pos_copybuf);
    thrust::device_ptr<float> charge_ptr(d_charge);
    thrust::device_ptr<float> charge_copybuf_ptr(d_charge_copybuf);
    thrust::device_ptr<float> diameter_ptr(d_diameter);
    thrust::device_ptr<float> diameter_copybuf_ptr(d_diameter_copybuf);
    thrust::device_ptr<unsigned char> plan_ptr(d_plan);
    thrust::device_ptr<unsigned char> plan_copybuf_ptr(d_plan_copybuf);

    permutation_iterator<device_ptr<unsigned int>, device_ptr<unsigned int> > ghost_rtag(rtag_ptr, copy_ghosts_ptr);
    gather(ghost_rtag, ghost_rtag + nghost, pos_ptr, pos_copybuf_ptr);
    gather(ghost_rtag, ghost_rtag + nghost, charge_ptr, charge_copybuf_ptr);
    gather(ghost_rtag, ghost_rtag + nghost, diameter_ptr, diameter_copybuf_ptr);
    gather(ghost_rtag, ghost_rtag + nghost, plan_ptr, plan_copybuf_ptr);
    }

//! Update global tag <-> local particle index reverse lookup array
/*! \param nptl Number of particles for which we are updating the reverse lookup tags
 * \param start_idx starting index of first particle in local particle data arrays
 * \param d_tag array of particle tags
 * \param d_rtag array of particle reverse lookup tags to store information to
 */
void gpu_update_rtag(unsigned int nptl, unsigned int start_idx, unsigned int *d_tag, unsigned int *d_rtag)
    {
    thrust::device_ptr<unsigned int> tag_ptr(d_tag);
    thrust::device_ptr<unsigned int> rtag_ptr(d_rtag);

    thrust::counting_iterator<unsigned int> first(start_idx);
    thrust::counting_iterator<unsigned int> last = first + nptl;
    thrust::scatter(first, last, tag_ptr, rtag_ptr);
    }

//! Copy ghost particle positions into send buffer
/*! \param nghost Number of ghost particles to copy
 * \param d_pos Array of particle positions
 * \param d_copy_ghosts Global particle tags of particles to copy
 * \param d_pos_copybuf Send buffer of ghost particle positions
 * \param d_rtag Global tag <-> local particle index reverse lookup array
 */
void gpu_copy_ghosts(unsigned int nghost,
                     float4 *d_pos,
                     unsigned int *d_copy_ghosts,
                     float4 *d_pos_copybuf,
                     unsigned int *d_rtag)
    {
    thrust::device_ptr<float4> pos_ptr(d_pos);
    thrust::device_ptr<unsigned int> rtag_ptr(d_rtag);
    thrust::device_ptr<unsigned int> copy_ghosts_ptr(d_copy_ghosts);
    thrust::device_ptr<float4> copybuf_ptr(d_pos_copybuf);

    permutation_iterator<device_ptr<unsigned int>, device_ptr<unsigned int> > ghost_rtag(rtag_ptr, copy_ghosts_ptr);
    gather(ghost_rtag, ghost_rtag + nghost, pos_ptr, copybuf_ptr);

    }

//! Reset reverse lookup tags of removed ghost particles to NOT_LOCAL
/*! \param nghost Number of ghost particles for which the tags are to be reset
 * \param d_gloal_rtag Pointer to reverse-lookup tags to reset
 */
void gpu_reset_ghost_rtag(unsigned int nghost,
                          unsigned int *d_global_rtag)
     {
     thrust::device_ptr<unsigned int> global_rtag_ptr(d_global_rtag);
     thrust::fill(global_rtag_ptr, global_rtag_ptr + nghost, NOT_LOCAL);
     }
#endif
